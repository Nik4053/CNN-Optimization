#include <stdio.h>
#include <iostream>
#include "errorchecks.cuh"
#include "hip/hip_runtime.h"



void checkLast(char const* const file, int const line)
{
    hipError_t err{ hipGetLastError() };
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
            << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}