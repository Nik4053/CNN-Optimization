#include "hip/hip_runtime.h"
#include "cb_implementations.cuh"

inline __device__ unsigned get_lane_id() {
	unsigned ret;
	asm volatile("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

template<int filter_size, int yBatch>
__global__
void cb_padded_norelu_reorder_work(Tensor<float, 4> errorPadded, Tensor<float, 4> next_error, Tensor<float,4> outputPadded,Tensor<float, 4> weights) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y * yBatch + threadIdx.y * yBatch;
	int c_in = blockIdx.z * blockDim.z;

	int batch_size = errorPadded.getDim(0), 
		input_channels = next_error.getDim(1),
		output_channels = errorPadded.getDim(1),
		height = next_error.getDim(2),
		width = next_error.getDim(3);
	// assert(batch_size = errorPadded.getDim(0));
	// assert(errorPadded.getDim(2)==width);
	if (x >= width || y >= height || c_in >= input_channels)
		return;

	float reg_weights[filter_size][filter_size];
	// float reg_vals[batch_size];

	// for (int b = 0; b < batch_size; b++) {
	// 	reg_vals[b] = 0.;
	// }

	for (int c_out = 0; c_out < output_channels; c_out++) {
		#pragma unroll
		for (int j = 0; j < filter_size; j++) {
			for (int i = 0; i < filter_size; i++) {
				reg_weights[j][i] = weights(c_out, c_in, filter_size - j - 1, filter_size - i - 1);
			}
		}
		// // Alternative weights loading implementation
		// {
		// 	int lane_id = get_lane_id() ;
		// 	int local_weight_pos = weights.pos(c_out, c_in, 0, 0);
		// 	float local_weight = 0;
		// 	int filter_size_2 = filter_size*filter_size;
		// 	if(lane_id < filter_size_2)
		// 		local_weight = weights(local_weight_pos + lane_id);
		// 	#define FULL_MASK 0xffffffff
		// 	// unsigned mask = __ballot_sync(FULL_MASK, warp_id < filter_size_2);
		// 	// for (int offset = filter_size_2/2; offset > 0; offset /= 2)
		// 	// 	local_weight += __shfl_down_sync(mask, local_weight, offset);
		// 	int offset = 0;
		// 	#pragma unroll
		// 	for (int j = 0; j < filter_size; j++) {
		// 		for (int i = 0; i < filter_size; i++) {
		// 			reg_weights[filter_size - j - 1][filter_size - i - 1] =  __shfl_sync(FULL_MASK, local_weight, offset);
		// 			offset++;
		// 		}
		// 	}
		// }
		#pragma unroll
		for ( int yy = 0; yy < yBatch; yy++){
			Debug(if (y + yy >= height) continue;) // check if we are in bounds
			for (int b = 0; b < batch_size; b++) {
				float val = 0.;
				for (int j = 0; j < filter_size; j++) {
					for (int i = 0; i < filter_size; i++) {
						float error_val = errorPadded(b, c_out, y + yy + j, x + i);
						val += error_val * reg_weights[j][i];
					}
				}
				next_error(b, c_in, y+ yy, x) += val;
			}
		}
	}

	
}
template __global__ void cb_padded_norelu_reorder_work<3,1>(Tensor<float, 4> error, Tensor<float, 4> next_error, Tensor<float,4> output,Tensor<float, 4> weights);
template __global__ void cb_padded_norelu_reorder_work<3,4>(Tensor<float, 4> error, Tensor<float, 4> next_error, Tensor<float,4> output,Tensor<float, 4> weights);
template __global__ void cb_padded_norelu_reorder_work<3,8>(Tensor<float, 4> error, Tensor<float, 4> next_error, Tensor<float,4> output,Tensor<float, 4> weights);
template __global__ void cb_padded_norelu_reorder_work<3,16>(Tensor<float, 4> error, Tensor<float, 4> next_error, Tensor<float,4> output,Tensor<float, 4> weights);