#include "hip/hip_runtime.h"
#include "implementations.cuh"



// reordered the for loops to get better memory access
template<int filter_size>
__global__
void w_shfl_registers_reordered_padded(Tensor<float, 4> inputPadded, Tensor<float, 4> error, Tensor<float, 4> gradient_weights) {
	const int block_y = blockIdx.y;
	const int block_x = blockIdx.x;
	const int thread_y = threadIdx.y;
	const int thread_x = threadIdx.x;

	// get our position in the grid. #blocks*blocksize + position in block
	const int x = block_x * blockDim.x  + thread_x;
	const int y = block_y * blockDim.y  + thread_y;
	const int z = blockIdx.z * blockDim.z;


	// asserts for debug
	// assert(input.getDim(3) == error.getDim(3) && input.getDim(2) == error.getDim(2)); // img size has to be identical 
	// assert(input.getDim(0) == error.getDim(0)); // batch size has to be identical
	// assert(input.getDim(1) == gradient_weights.getDim(1)); // input channels have to be identical
	// assert(error.getDim(1) == gradient_weights.getDim(0)); // output channels have to be identical
	// assert(gradient_weights.getDim(2) == filter_size && gradient_weights.getDim(3) == filter_size); // filter size has to be identical



	const int batch_size = inputPadded.getDim(0),
				input_channels = inputPadded.getDim(1),
				output_channels = error.getDim(1),
				height = error.getDim(2),
				width = error.getDim(3);

	// assert(x < width && y < height && z < gradient_weights.getDim(0) * gradient_weights.getDim(1)); // check if we are in bounds

	// get the channels we are working on
	const int c_in = z % input_channels;
	const int c_out = z / input_channels;

	// local id for shared memory
	const int tid = thread_y * blockDim.x + thread_x;
	// const int threads_in_block = blockDim.x * blockDim.y;
	
	assert(c_in < input_channels && c_out < output_channels);
   

	

	extern __shared__ float sm[];
	float val[filter_size][filter_size];

	// init val
	for(int j=0;j<filter_size;j++){
		for(int i=0;i<filter_size;i++){
			val[j][i]=0;
		}
	}
	for (int b = 0; b < batch_size; b++) {
		#pragma unroll
		for (int j = 0; j < filter_size; j++) {
			for (int i =0; i < filter_size; i++) {
				val[j][i] += inputPadded(b, c_in, y + j, x + i) * error(b, c_out, y, x);	
			}
		}
	}
	
	sm[tid] = 0;
	__syncthreads();
	for (int j = 0; j < filter_size; j++) {
		for (int i =0; i < filter_size; i++) {
			
			double vval = val[j][i];
			for (int offset = 16; offset > 0; offset /= 2)
				vval += __shfl_down_sync(0xffffffff, vval, offset);
			if (tid % 32 == 0) {
				sm[tid/32] = vval;
			}
			__syncthreads();

			// int n = threads_in_block;
			if (tid < 32) { // only works if threads_in_block/32 <= 32 or threads_in_block <= 1024
				double val = sm[tid];
				for (int offset = 16; offset > 0; offset /= 2)
					val += __shfl_down_sync(0xffffffff, val, offset);
				
				if (tid == 0)
					atomicAdd(&gradient_weights(c_out, c_in, j, i), val);
			}
		}
	}
}
template __global__ void  w_shfl_registers_reordered_padded<3>(Tensor<float, 4> inputPadded, Tensor<float, 4> error, Tensor<float, 4> gradient_weights);