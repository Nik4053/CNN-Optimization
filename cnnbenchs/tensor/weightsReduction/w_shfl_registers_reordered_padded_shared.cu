#include "hip/hip_runtime.h"
#include "implementations.cuh"



// reordered the for loops to get better memory access
template<int filter_size, int batch_size>
__global__
void w_shfl_registers_reordered_padded_shared(Tensor<float, 4> inputPadded, Tensor<float, 4> error, Tensor<float, 4> gradient_weights) {
	const short block_y = blockIdx.y;
	const short block_x = blockIdx.x;
	const short thread_y = threadIdx.y;
	const short thread_x = threadIdx.x;

	// get our position in the grid. #blocks*blocksize + position in block
	const short x = block_x * blockDim.x  + thread_x;
	const short y = block_y * blockDim.y  + thread_y;
	const short z = blockIdx.z * blockDim.z;


	// asserts for debug
	// assert(input.getDim(3) == error.getDim(3) && input.getDim(2) == error.getDim(2)); // img size has to be identical 
	// assert(input.getDim(0) == error.getDim(0)); // batch size has to be identical
	// assert(input.getDim(1) == gradient_weights.getDim(1)); // input channels have to be identical
	// assert(error.getDim(1) == gradient_weights.getDim(0)); // output channels have to be identical
	// assert(gradient_weights.getDim(2) == filter_size && gradient_weights.getDim(3) == filter_size); // filter size has to be identical
	assert(batch_size == inputPadded.getDim(0)); // batch size has to be identical


	const short input_channels = inputPadded.getDim(1),
				output_channels = error.getDim(1),
				height = error.getDim(2),
				width = error.getDim(3);

	// assert(x < width && y < height && z < gradient_weights.getDim(0) * gradient_weights.getDim(1)); // check if we are in bounds

	// get the channels we are working on
	const short c_in = z % input_channels;
	const short c_out = z / input_channels;

	// local id for shared memory
	const short tid = thread_y * blockDim.x + thread_x;
	// const short threads_in_block = blockDim.x * blockDim.y;
	
	assert(c_in < input_channels && c_out < output_channels);
   

	

	extern __shared__ float sm[];
	float val[filter_size][filter_size];
	float e_val[batch_size];

	// init val
	for (short j = 0; j < filter_size; j++) {
		for (short i =0; i < filter_size; i++) {
			val[j][i] = 0;
		}
	}

	for (short b = 0; b < batch_size; b++) {
		e_val[b] = error(b, c_out, y, x);
	}

	for (short b = 0; b < batch_size; b++) {
		#pragma unroll
		for (short j = 0; j < filter_size; j++) {
			for (short i =0; i < filter_size; i++) {
				val[j][i] += inputPadded(b, c_in, y + j, x + i) * e_val[b];	
			}
		}
	}
	sm[tid] = 0;
	__syncthreads();
	for (int j = 0; j < filter_size; j++) {
		for (int i =0; i < filter_size; i++) {
			
			double vval = val[j][i];
			for (int offset = 16; offset > 0; offset /= 2)
				vval += __shfl_down_sync(0xffffffff, vval, offset);
			if (tid % 32 == 0) {
				sm[tid/32] = vval;
			}
			__syncthreads();

			// int n = threads_in_block;
			if (tid < 32) { // only works if threads_in_block/32 <= 32 or threads_in_block <= 1024
				double val = sm[tid];
				for (int offset = 16; offset > 0; offset /= 2)
					val += __shfl_down_sync(0xffffffff, val, offset);
				
				if (tid == 0)
					atomicAdd(&gradient_weights(c_out, c_in, j, i), val);
			}
		}
	}
}
template __global__ void  w_shfl_registers_reordered_padded_shared<3,1>(Tensor<float, 4> inputPadded, Tensor<float, 4> error, Tensor<float, 4> gradient_weights);
template __global__ void  w_shfl_registers_reordered_padded_shared<3,10>(Tensor<float, 4> inputPadded, Tensor<float, 4> error, Tensor<float, 4> gradient_weights);
template __global__ void  w_shfl_registers_reordered_padded_shared<3,100>(Tensor<float, 4> inputPadded, Tensor<float, 4> error, Tensor<float, 4> gradient_weights);